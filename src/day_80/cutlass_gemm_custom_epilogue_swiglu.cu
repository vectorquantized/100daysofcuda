#include <hip/hip_runtime.h>
#include "../cuda/cutlass/gemm.h"
#include "../cuda/cutlass/util.h"
#include "../cuda/cuda_utils.h"
#include "../csrc/utils.h"

int main(int argc, char* argv[]) {

    int M = 1024;
    int N = 4096;
    int K = 2048;

    float alpha = 1.0f;
    float beta = 0.0;

    auto init_policy_A = std::make_unique<UniformInitPolicyKernel<cutlass::half_t, cutlass::layout::RowMajor>>(
        2080, -4.0_hf, 4.0_hf, 2);
        
    auto init_policy_B = std::make_unique<UniformInitPolicyKernel<cutlass::half_t, cutlass::layout::RowMajor>>(
        2081, -4.0_hf, 4.0_hf, 2);
        
    auto init_policy_C = std::make_unique<UniformInitPolicyKernel<cutlass::half_t, cutlass::layout::RowMajor>>(
        2082, -4.0_hf, 4.0_hf, 2);

    constexpr int Count = 128 / cutlass::sizeof_bits<cutlass::half_t>::value;
    using EpilogueSwiGlu = LinearCombinationSwiglu<
        cutlass::half_t, 
        Count,
        cutlass::Array<float, 2 * Count>, // ElementAccumulator_ (2×Count)
        float                     // ElementCompute_
    >;
    using AmpereSwigluConfig = GemmConfigWithEpilogue<cutlass::arch::Sm80, EpilogueSwiGlu>;
    cutlass::Status status = run_gemm_with_activation<float, AmpereSwigluConfig>(
        M, N, K, 
        alpha, beta,
        init_policy_A.get(),
        init_policy_B.get(),
        init_policy_C.get()
    );
    
    CUTLASS_CHECK(status);
    return 0;
}