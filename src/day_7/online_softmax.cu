#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <cfloat>
#include "csrc/utils.h"
#include "cpu/kernels.h"
#include "cuda/cuda_utils.h"
#include "cuda/tensor.h"

#define NUM_THREADS 256
#define TILE_WIDTH 32
#define EPSILON 1e-7


__global__ void online_softmax(ten::Tensor input, ten::Tensor output, size_t M, size_t N) {
    int row = threadIdx.x + blockIdx.x * blockDim.x;
    
    if (row < M) {

        float thread_max = -FLT_MAX;
        float norm = 0.0f;

        for (int col = 0; col < N; ++col) {
            float curr_value = input[row * N + col];
            if (curr_value > thread_max) {
                norm *= expf(thread_max - curr_value);
                thread_max = curr_value;
            }
            norm += expf(curr_value - thread_max);

        }

        for (int col = 0; col < N; ++col) {
            output[row * N + col] = expf(input[row * N + col] - thread_max) / (norm + EPSILON);
        }
    }
}

__global__ void softmax(ten::Tensor input, ten::Tensor output, size_t M, size_t N) {
    int row = threadIdx.x + blockIdx.x * blockDim.x;

    if (row < M) {
        float max_value = 0.0f;
        float row_sum = 0.0f;

        for (int col = 0; col < N; ++col) {
            max_value = max(max_value, input[row * N + col]);
        }

        for (int col = 0; col < N; ++col) {
            float value = expf(input[row * N + col] - max_value);
            row_sum += value;
        }

        for (int col = 0; col < N; ++col) {
            output[row * N + col] = expf(input[row * N + col] - max_value) / row_sum;
        }
    }
}

void kernel_launch(const ten::Tensor& input, const ten::Tensor& output, size_t M, size_t N) {

    dim3 threads_per_block(NUM_THREADS);
    dim3 blocks_per_grid((M + NUM_THREADS - 1) / NUM_THREADS);
    
    online_softmax<<<blocks_per_grid, threads_per_block>>>(input, output, M, N);
}

void kernel_launch_softmax(const ten::Tensor& input, const ten::Tensor& output, size_t M, size_t N) {

    dim3 threads_per_block(NUM_THREADS);
    dim3 blocks_per_grid((M + NUM_THREADS - 1) / NUM_THREADS);
    
    softmax<<<blocks_per_grid, threads_per_block>>>(input, output, M, N);
}

int main(int argc, char* argv[]) {
    size_t M = 8192;
    size_t N = 2048;

    unsigned int baseSeed = 42;
    // Use pinned memory for std::vector
    PinnedVector<float> a_h(M * N);
    PinnedVector<float> b_h(M * N);

    cpu_utils::init_random_vector(a_h, M * N, baseSeed);
    
    ten::Tensor a_d, b_d;
    {
        TIMED_CUDA_BLOCK("💾 Memory Allocation on Device");
        a_d.allocate(M * N);
        b_d.allocate(M * N);
    }
    
    // Memcpy HostToDevice
    {
        TIMED_CUDA_BLOCK("💾 Mem copy (hipMemcpyHostToDevice)");
        CUDA_ERROR_CHECK(hipMemcpy(a_d.data, a_h.data(), M * N * sizeof(float), hipMemcpyHostToDevice));
    }

    // {
    //     TIMED_CUDA_BLOCK("🚀 Kernel execution time");
    //     kernel_launch_softmax(a_d, b_d, M, N);
    //     CUDA_ERROR_CHECK(hipDeviceSynchronize()); // Barrier sync
    // }

    // {
    //     TIMED_CUDA_BLOCK("🚀 Online Kernel execution time");
    //     kernel_launch(a_d, b_d, M, N);
    //     CUDA_ERROR_CHECK(hipDeviceSynchronize()); // Barrier sync
    // }

    // {
    //     TIMED_CUDA_BLOCK("🚀 Kernel execution time");
    //     kernel_launch_softmax(a_d, b_d, M, N);
    //     CUDA_ERROR_CHECK(hipDeviceSynchronize()); // Barrier sync
    // }

    {
        TIMED_CUDA_BLOCK("🚀 Online Kernel execution time");
        kernel_launch(a_d, b_d, M, N);
        CUDA_ERROR_CHECK(hipDeviceSynchronize()); // Barrier sync
    }

    {
        TIMED_CUDA_BLOCK("💾 Mem copy (hipMemcpyDeviceToHost)");
        CUDA_ERROR_CHECK(hipMemcpy(b_h.data(), b_d.data, M * N * sizeof(float), hipMemcpyDeviceToHost));
    }

    a_d.free();
    b_d.free();
    
    std::vector<float> b_ref(a_h.size());
    b_ref.resize(a_h.size());
    cpu_kernels::online_softmax<float>(a_h, b_ref, M, N, EPSILON);
    COMPARE_RESULT(b_ref.data(), b_h.data(), M * N, 1e-5);
    return 0;
}