#include <hip/hip_runtime.h>
#include "../cuda/cutlass/gemm.h"
#include "../cuda/cuda_utils.h"
#include "../csrc/utils.h"

int main(int argc, char* argv[]) {

    hipDeviceProp_t deviceProp;
    hipError_t err = hipGetDeviceProperties(&deviceProp, 0);  // device 0
    if (err != hipSuccess) {
        std::cerr << "Error getting device properties: " << hipGetErrorString(err) << std::endl;
        return -1;
    }
    std::cout << "Compute capability: " 
              << deviceProp.major << "." << deviceProp.minor << std::endl;


    int M = 1024;
    int N = 512;
    int K = 256;
    int batch_count = 8;
    int kRange = 17;

    float alpha = 1.0f;
    float beta = 0.0;
    int lda = M;
    int ldb = K;
    int ldc = M;

    int count_A = batch_count * lda * K;
    int count_B = batch_count * ldb * N;
    int count_C = batch_count * ldc * N;

    std::vector<float> host_A(count_A);
    std::vector<float> host_B(count_B);
    std::vector<float> host_C(count_C);

    auto init_A = [kRange, lda, K] (int batch, int row, int col) -> float {
        return static_cast<float>((batch * lda * K + col * lda + row) % kRange);
    };
    
    auto init_B = [kRange, N, K, ldb, batch_count](int batch, int row, int col) -> float {
        return static_cast<float>(((N + K * ldb + batch_count * ldb * K)-(batch * ldb * K + col * ldb + row)) % kRange);
    };
    
    auto init_C = [](int batch, int row, int col) -> float {
        return 1.0f;
    };
    
    cpu_utils::initialize_batched_matrices_col_major(host_A.data(), batch_count, M, K, lda, lda * K, init_A);
    cpu_utils::initialize_batched_matrices_col_major(host_B.data(), batch_count, K, N, ldb, ldb * N, init_B);
    cpu_utils::initialize_batched_matrices_col_major(host_C.data(), batch_count, M, N, ldc, ldc * N, init_C);

    float* A;
    CUDA_ERROR_CHECK(hipMalloc(&A, count_A * sizeof(float)));
    CUDA_ERROR_CHECK(hipMemcpy(A, host_A.data(), count_A * sizeof(float), hipMemcpyHostToDevice));
    float* B;
    CUDA_ERROR_CHECK(hipMalloc(&B, count_B * sizeof(float)));
    CUDA_ERROR_CHECK(hipMemcpy(B, host_B.data(), count_B * sizeof(float), hipMemcpyHostToDevice));
    float* C;
    CUDA_ERROR_CHECK(hipMalloc(&C, count_C * sizeof(float)));
    CUDA_ERROR_CHECK(hipMemcpy(C, host_C.data(), count_C * sizeof(float), hipMemcpyHostToDevice));

    // Data has been copied over. We now allocate the memory for array of pointers.
    std::vector<float*> host_ptr_A(batch_count);
    std::vector<float*> host_ptr_B(batch_count);
    std::vector<float*> host_ptr_C(batch_count);

    // permute the batch elements to emphasize that GemmArray does not depend on matrices being separated by a fixed stride
    auto permutation = cpu_utils::generate_random_permutation(batch_count);
    for (size_t b_idx = 0; b_idx < batch_count; b_idx++) {
      host_ptr_A[b_idx] = A + permutation[b_idx] * lda * K;
      host_ptr_B[b_idx] = B + permutation[b_idx] * ldb * N;
      host_ptr_C[b_idx] = C + permutation[b_idx] * ldc * N;
    }

    // we copy these pointers now to device and we are ready!
    float const **ptr_A;
    CUDA_ERROR_CHECK(hipMalloc(&ptr_A, batch_count * sizeof(float*))); // we allocate an array of pointers
    CUDA_ERROR_CHECK(hipMemcpy(ptr_A, host_ptr_A.data(), batch_count * sizeof(float*), hipMemcpyHostToDevice));

    float const **ptr_B;
    CUDA_ERROR_CHECK(hipMalloc(&ptr_B, batch_count * sizeof(float*))); // we allocate an array of pointers
    CUDA_ERROR_CHECK(hipMemcpy(ptr_B, host_ptr_B.data(), batch_count * sizeof(float*), hipMemcpyHostToDevice));

    float **ptr_C;
    CUDA_ERROR_CHECK(hipMalloc(&ptr_C, batch_count * sizeof(float*))); // we allocate an array of pointers
    CUDA_ERROR_CHECK(hipMemcpy(ptr_C, host_ptr_C.data(), batch_count * sizeof(float*), hipMemcpyHostToDevice));

    cutlass::Status status;
    if(deviceProp.major == 8) {
        using AmpereConfig = GemmConfig<cutlass::arch::Sm80>;
        status = run_gemm_batched_array<float>(M, N, K, alpha, ptr_A, lda, ptr_B, ldb, ptr_C, ldc, beta, batch_count);
    } else if (deviceProp.major ==7 && deviceProp.minor == 5) {
        using TuringConfig = GemmConfig<cutlass::arch::Sm75>;
        status = run_gemm_batched_array<float>(M, N, K, alpha, ptr_A, lda, ptr_B, ldb, ptr_C, ldc, beta, batch_count);
    } else {
        std::cerr << "Unsupported compute capability: "
                  << deviceProp.major << "." << deviceProp.minor << std::endl;
        return -1;
    }
    
    hipError_t cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(cudaStatus) << std::endl;
    }

    if(status != cutlass::Status::kSuccess) {
        std::cout << "GEMM ERROR: " << static_cast<int>(status) << std::endl;
        return -1;
    }

    hipFree(A);
    hipFree(B);
    hipFree(C);
    hipFree(ptr_A);
    hipFree(ptr_B);
    hipFree(ptr_C);

    return 0;
}