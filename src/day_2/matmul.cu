#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include "csrc/utils.h"
#include "cpu/kernels.h"
#include "cuda/cuda_utils.h"
#include "cuda/tensor.h"


__global__ void gemm(ten::Tensor a, ten::Tensor b, ten::Tensor c, int M, int K, int N) {
    int row = threadIdx.y  + blockIdx.y * blockDim.y;
    int col = threadIdx.x + blockIdx.x * blockDim.x; 
    if (row < M && col < N) {
        float p_value = 0.0f;
        for(int i=0; i<K; ++i) {
            p_value += a[row * K + i] * b[i * N + col];
        }
        c[row * N + col] = p_value;
    }
}

void kernel_launch(const ten::Tensor& a_d, const ten::Tensor& b_d, ten::Tensor& c_d, size_t M, size_t K, size_t N) {
    TIMED_CUDA_FUNCTION();
    int block_size_x = 16;
    int block_size_y = 16;

    dim3 threads_per_block(block_size_x, block_size_y);
    dim3 blocks_per_grid (
                                (N + block_size_x - 1) / block_size_x, // output has N columns
                                (M + block_size_y - 1) / block_size_y // output has M rows
                            );
    gemm<<<blocks_per_grid, threads_per_block>>>(a_d, b_d, c_d, M, K, N);

    CUDA_ERROR_CHECK(hipDeviceSynchronize()); // Barrier sync
}

int main(int argc, char* argv[]) {
    size_t M = 1024; 
    size_t K = 1024;
    size_t N = 2048;

    unsigned int baseSeed = 42;
    std::vector<float> a_h(M * K);
    std::vector<float> b_h(K * N);
    std::vector<float> c_h(M * N);
    cpu_utils::init_random_vector(a_h, M * K, baseSeed);
    cpu_utils::init_random_vector(b_h, K * N, baseSeed + 1);
    ten::Tensor a_d, b_d, c_d;
    a_d.allocate(M * K);
    b_d.allocate(K * N);
    c_d.allocate(M * N);
    CUDA_ERROR_CHECK(hipMemcpy(a_d.data, a_h.data(), M * K * sizeof(float), hipMemcpyHostToDevice));
    CUDA_ERROR_CHECK(hipMemcpy(b_d.data, b_h.data(), K * N * sizeof(float), hipMemcpyHostToDevice));
    kernel_launch(a_d, b_d, c_d, M, K, N);
    CUDA_ERROR_CHECK(hipMemcpy(c_h.data(), c_d.data, M * N * sizeof(float), hipMemcpyDeviceToHost));
    a_d.free();
    b_d.free();
    c_d.free();
    std::vector<float> c_ref(M * N);
    cpu_kernels::gemm(a_h, b_h, c_ref, M, K, N);
    COMPARE_RESULT(c_ref.data(), c_h.data(), M*N, 1e-3);
    return 0;
}