#include <hip/hip_runtime.h>
#include "../cuda/cutlass/gemm.h"
#include "../cuda/cuda_utils.h"
#include "../csrc/utils.h"
#include <iostream>
#include <vector>
#include <algorithm>


int main(int argc, char* argv[]) {

    hipDeviceProp_t deviceProp;
    hipError_t err = hipGetDeviceProperties(&deviceProp, 0);  // device 0
    if (err != hipSuccess) {
        std::cerr << "Error getting device properties: " << hipGetErrorString(err) << std::endl;
        return -1;
    }
    std::cout << "Compute capability: " 
              << deviceProp.major << "." << deviceProp.minor << std::endl;


    int B = 32;   // batch size
    int N = 8;    // heads
    int G = 4;    // number of key/value groups
    int L = 128;  // sequence length
    int D = 64;   // head dimension
    
    int ldq = L;
    int ldK = L;

    int count_Q = B * N * L * D;
    int count_K = B * G * L * D;
    int count_Scores = B * N * L * L;

    std::vector<float> host_Q(count_Q);
    std::vector<float> host_K(count_K);
    std::vector<float> host_Scores(count_Scores);

    // Allocate and init V and Output
    int count_V = B * G * L * D;
    int count_Output = B * N * L * D;
    std::vector<float> host_V(count_V);
    std::vector<float> host_Output(count_Output);

    int kRange = 100;

    auto init_Q = [kRange, N, ldq, D] (int batch, int head, int row, int col) -> float {
        return static_cast<float>((batch * N * ldq * D + head * ldq * D + col * ldq + row) % kRange);
    };
    
    auto init_K = [kRange, ldK, D](int batch, int head, int row, int col) -> float {
        return static_cast<float>((batch * ldK * D + col * ldK + row) % kRange);
    };
    
    auto init_Scores = [](int batch, int head, int row, int col) -> float {
        return 1.0f;
    };
    
    cpu_utils::initialize_batched_multi_headed_matrices_col_major(host_Q.data(), B * N, N, L, D, L, L * D, init_Q);
    cpu_utils::initialize_batched_multi_headed_matrices_col_major(
        host_K.data(), B * G, G, L, D, L, L * D, init_K);
    cpu_utils::initialize_batched_multi_headed_matrices_col_major(host_Scores.data(), B * N, N, L, L, L, L * L, init_Scores);

    // Initialize V same as K
    cpu_utils::initialize_batched_multi_headed_matrices_col_major(
        host_V.data(), B * G, G, L, D, L, L * D, init_K);
    // Initialize Output to zero
    std::fill(host_Output.begin(), host_Output.end(), 0.0f);

    float* d_Q;
    CUDA_ERROR_CHECK(hipMalloc(&d_Q, count_Q * sizeof(float)));
    CUDA_ERROR_CHECK(hipMemcpy(d_Q, host_Q.data(), count_Q * sizeof(float), hipMemcpyHostToDevice));
    float* d_K;
    CUDA_ERROR_CHECK(hipMalloc(&d_K, count_K * sizeof(float)));
    CUDA_ERROR_CHECK(hipMemcpy(d_K, host_K.data(), count_K * sizeof(float), hipMemcpyHostToDevice));
    float* d_Scores;
    CUDA_ERROR_CHECK(hipMalloc(&d_Scores, count_Scores * sizeof(float)));
    CUDA_ERROR_CHECK(hipMemcpy(d_Scores, host_Scores.data(), count_Scores * sizeof(float), hipMemcpyHostToDevice));

    float* d_V;
    CUDA_ERROR_CHECK(hipMalloc(&d_V, count_V * sizeof(float)));
    CUDA_ERROR_CHECK(hipMemcpy(d_V, host_V.data(), count_V * sizeof(float), hipMemcpyHostToDevice));

    float* d_Output;
    CUDA_ERROR_CHECK(hipMalloc(&d_Output, count_Output * sizeof(float)));
    CUDA_ERROR_CHECK(hipMemcpy(d_Output, host_Output.data(), count_Output * sizeof(float), hipMemcpyHostToDevice));

    cutlass::Status status;
    
    status = run_gqa_looped<float>(B, N, G, L, D, d_Q, d_K, d_V, d_Scores, d_Output);
    
    hipError_t cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(cudaStatus) << std::endl;
    }

    if(status != cutlass::Status::kSuccess) {
        std::cout << "GEMM ERROR: " << static_cast<int>(status) << std::endl;
        return -1;
    }

    hipFree(d_Q);
    hipFree(d_K);
    hipFree(d_Scores);
    hipFree(d_V);
    hipFree(d_Output);

    return 0;
}