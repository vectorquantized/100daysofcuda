#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include "csrc/utils.h"
#include "cpu/kernels.h"
#include "cuda/cuda_utils.h"
#include "cuda/tensor.h"

#define NUM_THREADS 256


__global__ void softmax(ten::Tensor input, ten::Tensor output, size_t M, size_t N) {
    int row = threadIdx.x + blockIdx.x * blockDim.x;

    if (row < M) {
        float max_value = 0.0f;
        float row_sum = 0.0f;

        for (int col = 0; col < N; ++col) {
            max_value = max(max_value, input[row * N + col]);
        }

        for (int col = 0; col < N; ++col) {
            float value = expf(input[row * N + col] - max_value);
            row_sum += value;
        }

        for (int col = 0; col < N; ++col) {
            output[row * N + col] = expf(input[row * N + col] - max_value) / row_sum;
        }
    }
}


void kernel_launch(const ten::Tensor& a_d, const ten::Tensor& b_d, size_t M, size_t N) {
    int block_size_x = NUM_THREADS;
    int block_size_y = NUM_THREADS;

    dim3 threads_per_block(block_size_x, block_size_y);
    dim3 blocks_per_grid ((M + block_size_y - 1) / block_size_y);
    softmax<<<blocks_per_grid, threads_per_block>>>(a_d, b_d, M, N);
}




int main(int argc, char* argv[]) {
    size_t M = 8192;
    size_t N = 4096;

    unsigned int baseSeed = 42;
    PinnedVector<float> a_h(M * N);
    PinnedVector<float> b_h(M * N);
    cpu_utils::init_random_vector(a_h, M * N, baseSeed);
    
    ten::Tensor a_d, b_d;
    a_d.allocate(M * N);
    b_d.allocate(M * N);
    
    {
        TIMED_CUDA_BLOCK("💾 Memory Allocation on Device");
        a_d.allocate(M * N);
        b_d.allocate(M * N);
    }
    
    {
        TIMED_CUDA_BLOCK("💾 Mem copy (hipMemcpyHostToDevice)");
        CUDA_ERROR_CHECK(hipMemcpy(a_d.data, a_h.data(), M * N * sizeof(float), hipMemcpyHostToDevice));
    }

    {
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);
        // TIMED_CUDA_BLOCK("🚀 Kernel execution time");
        kernel_launch(a_d, b_d, M, N);
        CUDA_ERROR_CHECK(hipDeviceSynchronize()); // Barrier sync
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        float milliseconds = 0.0f;
        hipEventElapsedTime(&milliseconds, start, stop);
        hipEventDestroy(start);
        hipEventDestroy(stop);
        constexpr size_t col_width = 40;
        
        std::cout << std::left << std::setw(col_width) <<  "🚀 Kernel execution time"
                  << ":  " << std::fixed << std::setprecision(3) 
                  << std::setw(8) << milliseconds << " ms" << std::endl;
    }

    {
        TIMED_CUDA_BLOCK("💾 Mem copy (hipMemcpyDeviceToHost)");
        CUDA_ERROR_CHECK(hipMemcpy(b_h.data(), b_d.data, M * N * sizeof(float), hipMemcpyDeviceToHost));
    }
    
    a_d.free();
    b_d.free();
    
    std::vector<float> b_ref = cpu_kernels::softmax<float>(a_h);
    COMPARE_RESULT(b_ref.data(), b_h.data(), M * N, 1e-6);
    cpu_utils::print_vectors(b_ref.data(), b_h.data(), M*N);
    return 0;
}