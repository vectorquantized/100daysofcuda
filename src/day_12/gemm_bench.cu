#include <vector>
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipblaslt.h>

torch::Tensor batched_gemm_forward(torch::Tensor A, torch::Tensor B, float scale) {
    TORCH_CHECK(A.device().is_cuda(), "A must be a CUDA tensor");
    TORCH_CHECK(B.device().is_cuda(), "B must be a CUDA tensor");
    TORCH_CHECK(A.ndimension() == 3 && B.ndimension() == 3, "A and B must be 3D tensors.");
    TORCH_CHECK(A.dtype() == torch::kFloat32, "A must be float32");
    TORCH_CHECK(B.dtype() == torch::kFloat32, "B must be float32");
    TORCH_CHECK(A.size(0) == B.size(0), "Batch sizes must match.");
    TORCH_CHECK(A.size(2) == B.size(1), "A's columns must match B's rows.");

    int batch_size = A.size(0);
    int M = A.size(1);
    int K = A.size(2);
    int N = B.size(2);

    auto C = torch::zeros({batch_size, M, N}, torch::TensorOptions().device(A.device()).dtype(A.dtype()));

    // Leading dimensions and strides
    int64_t lda = K;
    int64_t ldb = N;
    int64_t ldc = N;
    int64_t stridea = M * K;
    int64_t strideb = K * N;
    int64_t stridec = M * N;

    // Create cuBLASLt handle
    hipblasLtHandle_t ltHandle;
    hipblasLtCreate(&ltHandle);

    // Create Matmul Descriptor
    hipblasLtMatmulDesc_t operationDesc;
    hipblasLtMatmulDescCreate(&operationDesc, HIPBLAS_COMPUTE_32F, HIP_R_32F);
    hipblasOperation_t transA = HIPBLAS_OP_N, transB = HIPBLAS_OP_N;
    hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &transA, sizeof(transA));
    hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &transB, sizeof(transB));

    // Create Matrix Layouts
    hipblasLtMatrixLayout_t Adesc, Bdesc, Cdesc;
    hipblasLtMatrixLayoutCreate(&Adesc, HIP_R_32F, M, K, lda);
    hipblasLtMatrixLayoutCreate(&Bdesc, HIP_R_32F, K, N, ldb);
    hipblasLtMatrixLayoutCreate(&Cdesc, HIP_R_32F, M, N, ldc);

    // Set the matrix order to row-major for each descriptor
    int32_t order = HIPBLASLT_ORDER_ROW;
    hipblasLtMatrixLayoutSetAttribute(Adesc, HIPBLASLT_MATRIX_LAYOUT_ORDER, &order, sizeof(order));
    hipblasLtMatrixLayoutSetAttribute(Bdesc, HIPBLASLT_MATRIX_LAYOUT_ORDER, &order, sizeof(order));
    hipblasLtMatrixLayoutSetAttribute(Cdesc, HIPBLASLT_MATRIX_LAYOUT_ORDER, &order, sizeof(order));

    // Set Batch Attributes
    hipblasLtMatrixLayoutSetAttribute(Adesc, HIPBLASLT_MATRIX_LAYOUT_BATCH_COUNT, &batch_size, sizeof(batch_size));
    hipblasLtMatrixLayoutSetAttribute(Bdesc, HIPBLASLT_MATRIX_LAYOUT_BATCH_COUNT, &batch_size, sizeof(batch_size));
    hipblasLtMatrixLayoutSetAttribute(Cdesc, HIPBLASLT_MATRIX_LAYOUT_BATCH_COUNT, &batch_size, sizeof(batch_size));

    hipblasLtMatrixLayoutSetAttribute(Adesc, HIPBLASLT_MATRIX_LAYOUT_STRIDED_BATCH_OFFSET, &stridea, sizeof(stridea));
    hipblasLtMatrixLayoutSetAttribute(Bdesc, HIPBLASLT_MATRIX_LAYOUT_STRIDED_BATCH_OFFSET, &strideb, sizeof(strideb));
    hipblasLtMatrixLayoutSetAttribute(Cdesc, HIPBLASLT_MATRIX_LAYOUT_STRIDED_BATCH_OFFSET, &stridec, sizeof(stridec));

    size_t workspace_size = 1 * 1024 * 1024;
    void* workspace;
    hipMalloc(&workspace, workspace_size);

    hipblasLtMatmulPreference_t preference;
    hipblasLtMatmulPreferenceCreate(&preference);
    hipblasLtMatmulPreferenceSetAttribute(
        preference, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES, 
        &workspace_size, sizeof(workspace_size));

    hipblasLtMatmulHeuristicResult_t heuristicResult[1]; 
    int returnedResults = 0;
    hipblasStatus_t status = hipblasLtMatmulAlgoGetHeuristic(
        ltHandle, operationDesc, Adesc, Bdesc, Cdesc, Cdesc, preference, 
        1, 
        heuristicResult, &returnedResults);

    if (status != HIPBLAS_STATUS_SUCCESS || returnedResults == 0) {
        std::cerr << "No suitable cuBLASLt algorithm found!\n";
        return torch::empty({}); 
    }

   
    float alpha = 1.0f;
    float beta = 0.0f;

    
    hipStream_t stream;
    hipStreamCreate(&stream);

    hipblasLtMatmul(
        ltHandle,
        operationDesc,
        &alpha,
        A.data_ptr<float>(), // A matrix
        Adesc,
        B.data_ptr<float>(), // B matrix
        Bdesc,
        &beta,
        C.data_ptr<float>(), // C matrix (input)
        Cdesc,
        C.data_ptr<float>(), // C matrix (output)
        Cdesc,
        &heuristicResult[0].algo,
        workspace,
        workspace_size,
        stream);

   
    hipStreamSynchronize(stream);


    hipblasLtMatmulPreferenceDestroy(preference);
    hipblasLtMatrixLayoutDestroy(Adesc);
    hipblasLtMatrixLayoutDestroy(Bdesc);
    hipblasLtMatrixLayoutDestroy(Cdesc);
    hipblasLtMatmulDescDestroy(operationDesc);
    hipblasLtDestroy(ltHandle);
    hipFree(workspace);
    hipStreamDestroy(stream);

    return C;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("bmm_fast", &batched_gemm_forward, "Batched CUBLAS GEMM forward pass");
}