#include <hip/hip_runtime.h>
#include "../cuda/cutlass/gemm.h"
#include "../cuda/cuda_utils.h"
#include "../csrc/utils.h"
#include "../cuda/activations.h"

#define TILE_WIDTH 16
#define CEIL_DIV(M, N) ((M + N - 1 ) / N)

int main(int argc, char* argv[]) {

    hipDeviceProp_t deviceProp;
    hipError_t err = hipGetDeviceProperties(&deviceProp, 0);  // device 0
    if (err != hipSuccess) {
        std::cerr << "Error getting device properties: " << hipGetErrorString(err) << std::endl;
        return -1;
    }
    std::cout << "Compute capability: " 
              << deviceProp.major << "." << deviceProp.minor << std::endl;


    // A: batch_count, M, K => 8, 128, 2048
    // B: K, 4K => 256, 2048
    // C: K, 4K => 256, 2048
    // up = A @ B: batch_count, M, 4K
    // gate = A @ C: batch_count, M, 4K
    // swiglu(A): up * gate * sigmoid(gate) => (A @ B) * (A @ C) * sigmoid(A @ C)
    int M = 128;
    int K = 256;
    int N = 4 * K;
    int batch_count = 8;
    int kRange = 17;

    float alpha = 1.0f;
    float beta = 0.0;
    int lda = M;
    int ldup = K;
    int ldgate = K;
    int ldout = M;
    int batch_stride_A = lda * K;
    int batch_stride_up = 0; // weights are shared across all batches.
    int batch_stride_gate = 0; // if I don't do this then I'd have to replicate up and gate weights.
    int batch_stride_out = ldout * N;

    int count_A = batch_count * lda * K;
    int count_up = 1 * ldup * N;
    int count_gate = 1 * ldgate * N;
    int count_y = batch_count * ldout * N;
    int count_z = batch_count * ldout * N;
    int count_out = batch_count * ldout * N;

    std::vector<float> host_A(count_A);
    std::vector<float> host_up(count_up);
    std::vector<float> host_gate(count_gate);
    std::vector<float> host_y(count_y);
    std::vector<float> host_z(count_z);
    std::vector<float> host_out(count_out);

    auto init_A = [kRange, lda, K] (int batch, int row, int col) -> float {
        return static_cast<float>((batch * lda * K + col * lda + row) % kRange);
    };
    
    auto init_up = [kRange, N, K, ldup](int batch, int row, int col) -> float {
        return static_cast<float>((batch * ldup * K  + ldup * K + col * ldup + row) % kRange);
    };

    auto init_gate = [kRange, N, K, ldgate](int batch, int row, int col) -> float {
        return static_cast<float>((batch * ldgate * K + ldgate * K + col * ldgate + row) % kRange);
    };
    
    auto init_out = [](int batch, int row, int col) -> float {
        return 1.0f;
    };

    
    cpu_utils::initialize_batched_matrices_col_major(host_A.data(), batch_count, M, K, lda, lda * K, init_A);
    cpu_utils::initialize_batched_matrices_col_major(host_up.data(), 1, K, N, ldup, ldup * N, init_up);
    cpu_utils::initialize_batched_matrices_col_major(host_gate.data(), 1, K, N, ldgate, ldgate * N, init_gate);
    cpu_utils::initialize_batched_matrices_col_major(host_y.data(), batch_count, M, N, ldout, ldout * N, init_out);
    cpu_utils::initialize_batched_matrices_col_major(host_z.data(), batch_count, M, N, ldout, ldout * N, init_out);
    cpu_utils::initialize_batched_matrices_col_major(host_out.data(), batch_count, M, N, ldout, ldout * N, init_out);

    float* A;
    CUDA_ERROR_CHECK(hipMalloc(&A, count_A * sizeof(float)));
    CUDA_ERROR_CHECK(hipMemcpy(A, host_A.data(), count_A * sizeof(float), hipMemcpyHostToDevice));
    float* up;
    CUDA_ERROR_CHECK(hipMalloc(&up, count_up * sizeof(float)));
    CUDA_ERROR_CHECK(hipMemcpy(up, host_up.data(), count_up * sizeof(float), hipMemcpyHostToDevice));
    float* gate;
    CUDA_ERROR_CHECK(hipMalloc(&gate, count_gate * sizeof(float)));
    CUDA_ERROR_CHECK(hipMemcpy(gate, host_gate.data(), count_gate * sizeof(float), hipMemcpyHostToDevice));
    float* y;
    CUDA_ERROR_CHECK(hipMalloc(&y, count_y * sizeof(float)));
    CUDA_ERROR_CHECK(hipMemcpy(y, host_y.data(), count_y * sizeof(float), hipMemcpyHostToDevice));
    float* z;
    CUDA_ERROR_CHECK(hipMalloc(&z, count_z * sizeof(float)));
    CUDA_ERROR_CHECK(hipMemcpy(z, host_z.data(), count_z * sizeof(float), hipMemcpyHostToDevice));
    float* out;
    CUDA_ERROR_CHECK(hipMalloc(&out, count_out * sizeof(float)));
    CUDA_ERROR_CHECK(hipMemcpy(out, host_out.data(), count_out * sizeof(float), hipMemcpyHostToDevice));

    cutlass::Status status_up, status_gate;
    status_up = run_gemm_batched<float>(M, N, K, alpha, A, lda, batch_stride_A, up, ldup, batch_stride_up, y, ldout, batch_stride_out, beta, batch_count);
    status_gate = run_gemm_batched<float>(M, N, K, alpha, A, lda, batch_stride_A, gate, ldgate, batch_stride_gate, z, ldout, batch_stride_out, beta, batch_count);

    hipError_t cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(cudaStatus) << std::endl;
    }

    if(status_up != cutlass::Status::kSuccess) {
        std::cout << "GEMM ERROR in up projections: " << static_cast<int>(status_up) << std::endl;
        return -1;
    }

    if(status_gate != cutlass::Status::kSuccess) {
        std::cout << "GEMM ERROR in gate projection: " << static_cast<int>(status_gate) << std::endl;
        return -1;
    }

    int num_elements = count_out;
    int block_size = TILE_WIDTH * TILE_WIDTH;
    dim3 block_dim(block_size);
    dim3 grid_dim(CEIL_DIV(num_elements, block_size));
    
    swiglu<float><<<grid_dim, block_dim>>>(y, z, out, batch_count, M, N);

    CUDA_ERROR_CHECK(hipGetLastError());
    CUDA_ERROR_CHECK(hipDeviceSynchronize());

    hipFree(A);
    hipFree(up);
    hipFree(gate);
    hipFree(y);
    hipFree(z);
    hipFree(out);

    return 0;
}