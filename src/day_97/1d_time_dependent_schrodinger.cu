#include <iostream>
#include <fstream>
#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>

#define N 1024
#define DX 0.1f
#define DT 0.001f
#define NSTEPS 500

__global__ void evolve(hipFloatComplex* psi_new, const hipFloatComplex* psi_old, const float* V, float dx, float dt) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    if (j <= 0 || j >= N - 1) return; // skip boundaries

    hipFloatComplex laplacian = hipCsubf(psi_old[j + 1], hipCaddf(hipCmulf(make_hipFloatComplex(2.0f, 0.0f), psi_old[j]), psi_old[j - 1]));
    laplacian = hipCdivf(laplacian, make_hipFloatComplex(dx * dx, 0.0f));

    hipFloatComplex potential = hipCmulf(make_hipFloatComplex(V[j], 0.0f), psi_old[j]);

    hipFloatComplex rhs = hipCsubf(laplacian, potential);
    rhs = hipCmulf(make_hipFloatComplex(0.0f, -1.0f), rhs);

    psi_new[j] = hipCaddf(psi_old[j], hipCmulf(make_hipFloatComplex(dt, 0.0f), rhs));
}

void write_density(const hipFloatComplex* h_psi, int step) {
    std::ofstream file("psi_" + std::to_string(step) + ".pgm");
    file << "P2\n" << N << " 1\n255\n";
    for (int j = 0; j < N; ++j) {
        float prob = hipCabsf(h_psi[j]);
        int intensity = (int)(255 * prob);
        file << intensity << " ";
    }
    file << "\n";
    file.close();
}

int main() {
    hipFloatComplex* d_psi1;
    hipFloatComplex* d_psi2;
    float* d_V;

    hipFloatComplex* h_psi = new hipFloatComplex[N];
    float* h_V = new float[N];

    for (int j = 0; j < N; ++j) {
        float x = (j - N / 2) * DX;
        float sigma = 1.0f;
        float k0 = 5.0f;
        float envelope = expf(-x * x / (2 * sigma * sigma));
        h_psi[j] = make_hipFloatComplex(envelope * cosf(k0 * x), envelope * sinf(k0 * x));
        h_V[j] = 0.0f;
    }

    hipMalloc(&d_psi1, N * sizeof(hipFloatComplex));
    hipMalloc(&d_psi2, N * sizeof(hipFloatComplex));
    hipMalloc(&d_V, N * sizeof(float));

    hipMemcpy(d_psi1, h_psi, N * sizeof(hipFloatComplex), hipMemcpyHostToDevice);
    hipMemcpy(d_V, h_V, N * sizeof(float), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(256);
    dim3 numBlocks((N + threadsPerBlock.x - 1) / threadsPerBlock.x);

    for (int step = 0; step < NSTEPS; ++step) {
        evolve<<<numBlocks, threadsPerBlock>>>(d_psi2, d_psi1, d_V, DX, DT);
        std::swap(d_psi1, d_psi2);

        if (step % 50 == 0) {
            hipMemcpy(h_psi, d_psi1, N * sizeof(hipFloatComplex), hipMemcpyDeviceToHost);
            write_density(h_psi, step);
            std::cout << "Saved psi_" << step << ".pgm\n";
        }
    }

    hipFree(d_psi1);
    hipFree(d_psi2);
    hipFree(d_V);
    delete[] h_psi;
    delete[] h_V;

    std::cout << "Simulation completed.\n";
    return 0;
}