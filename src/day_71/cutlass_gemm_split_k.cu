#include <hip/hip_runtime.h>
#include "../cuda/cutlass/gemm.h"
#include "../cuda/cutlass/util.h"
#include "../cuda/cuda_utils.h"
#include "../csrc/utils.h"

int main(int argc, char* argv[]) {

    int M = 5120;
    int N = 4096;
    int K = 4096;

    float alpha = 1.0f;
    float beta = 0.0;
    int split_k_slices = 16;

    auto init_policy_A = std::make_unique<UniformInitPolicy<cutlass::half_t, cutlass::layout::ColumnMajor>>(
        2080, -4.0_hf, 4.0_hf, 2);
        
    auto init_policy_B = std::make_unique<UniformInitPolicy<cutlass::half_t, cutlass::layout::RowMajor>>(
        2081, -4.0_hf, 4.0_hf, 2);
        
    auto init_policy_C = std::make_unique<UniformInitPolicy<cutlass::half_t, cutlass::layout::RowMajor>>(
        2082, -4.0_hf, 4.0_hf, 2);

    using TiledAmpereConfig = TiledGemmConfig<cutlass::arch::Sm80>;
    cutlass::Status status = run_gemm_split_k<float, TiledAmpereConfig>(
        M, N, K, 
        alpha, beta, split_k_slices,
        init_policy_A.get(),
        init_policy_B.get(),
        init_policy_C.get()
    );
    
    CUTLASS_CHECK(status);
    return 0;
}